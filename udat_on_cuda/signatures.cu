#include <cstring>
#include <cstdio>
#include <fstream>



#include "file_manip.h"
#include "textures/zernike/zernike.h"
#include "haarlick.h"
#include "signatures.h"
#include "hip/hip_runtime.h"



Signatures::Signatures()
: row_len(INIT_MATRIX_CONTAINER_SIZE), 
  col_len(INIT_SIG_CONTAINER_SIZE)
{
  sigs = new char*[col_len];
  files = new char*[row_len];
  values = new double[row_len * col_len];
}



Signatures::~Signatures()
{ 
  clear();
}



void Signatures::add_signature(const char *sig_name, const char *filename, double value)
{
  int col = get_signature_index(sig_name);
  if(col < 0)
    col = insert_new_signature(sig_name);

  int row = get_filename_index(filename);
  if(row < 0)
    row = insert_new_filename(filename);

  values[row * col_len + col] = value;
}



double Signatures::get_signature(const char *sig_name, const char *filename) const
{
  return get_signature(get_signature_index(sig_name),
                       get_filename_index (filename));
}



double Signatures::get_signature(int col, const char *filename) const
{
  return get_signature(col, get_filename_index(filename));
}



double Signatures::get_signature(const char *sig_name, int row) const
{
  return get_signature(get_signature_index(sig_name), row);
}



double Signatures::get_signature(int col, int row) const
{
  if(col < 0 || col >= col_len || row < 0 || row >= row_len)
    return NAN;

  return values[row * col_len + col];
}



int Signatures::get_signature_index(const char *name) const
{
  return find_in_array(sigs, col_n, name);
}



int Signatures::get_filename_index(const char *name) const
{
  return find_in_array(files, row_n, name);
}



int Signatures::find_in_array(char **arr, int len, const char *element) const
{
  for(int i = 0; i < len; i++)
    if(strcmp(arr[i], element) == 0)
      return i;
  return -1;
}



void Signatures::clear()
{
  for(int i = 0; i < col_len; i++)
  {
    delete [] sigs[i];
    sigs[i] = 0;
  }

  for(int i = 0; i < row_len; i++)
  {
    delete [] files[i];
    files[i] = 0;
  }

  std::fill_n(values, row_len * col_len, NAN);
}



std::vector<std::string> Signatures::get_sig_names() const
{
  return get_array_copy(sigs, col_n);
}



std::vector<std::string> Signatures::get_filenames() const
{
  return get_array_copy(files, row_n);
}



int Signatures::insert_new_signature(const char *name) 
{
  if(col_n >= col_len)
    expand_signature_container();

  for(int i = 0; i < row_n; i++)
    values[i * col_len + col_n] = NAN;

  sigs[col_n] = new char[SIGNATURE_NAME_LENGTH];
  strcpy(sigs[col_n], name);

  col_n++;
  return col_n - 1;
}



char **expand_array(char **arr, int len, int new_len)
{
  char **new_arr = new char*[new_len];
  memset(new_arr, 0, new_len * sizeof(char *));

  for(int i = 0; i < len; i++) 
    new_arr[i] = arr[i];

  return new_arr;
}



void Signatures::expand_signature_container()
{
  int new_size = col_len * 2;
  char **new_sigs = expand_array(sigs, col_len, new_size);
  delete [] sigs;
  sigs = new_sigs;

  expand_value_array(row_len, new_size);
}



void Signatures::expand_filename_container()
{
  int new_size = row_len * 2;
  char **new_files = expand_array(files, row_len, new_size);
  delete [] files;
  files = new_files;

  expand_value_array(new_size, col_len);
}



void Signatures::expand_value_array(int d_row_len, int d_col_len)
{
  double *new_values = new double[d_row_len * d_col_len];
  std::fill_n(new_values, d_row_len * d_col_len, NAN);
  for(int row = 0; row < row_len; row++)
    for(int col = 0; col < col_len; col++)
      new_values[row * d_col_len + col] = values[row * col_len + col];
  delete [] values;
  col_len = d_col_len;
  row_len = d_row_len;
  values = new_values;
}



inline std::vector<std::string> Signatures::get_array_copy(char **arr, int len) const
{
  std::vector<std::string> arrcopy(len);
  for (int i = 0; i < len; i++)
    arrcopy.push_back(std::string(arr[i]));
  return arrcopy;
}



CUDASignatures::CUDASignatures()
:image_matrices(0), 
 matrix_container_size(INIT_MATRIX_CONTAINER_SIZE),
 image_matrix_count(0)
{ 
  image_matrices = new ImageMatrix*[matrix_container_size];
}



CUDASignatures::~CUDASignatures()
{
  empty_matrix_container();
  delete [] image_matrices;
}



void CUDASignatures::save_in(char *directory)
{
  char buffer[FILENAME_MAX];
  join_paths(buffer, directory, "output.csv");

  std::ofstream output(buffer);
  if (!output.good())
  {
    printf("Failed to open file \"%s\"", buffer);
    return;
  }

  std::vector<std::string> signature_names = signatures.get_sig_names();
  std::vector<std::string> filenames       = signatures.get_filenames();

  output << "filename";
  for(int i = 0; i < signature_names.size(); i++)
    output << ',' << signature_names[i]; 
  output << std::endl;

  for(std::string filename: filenames)
  {
    output << filename;

    for(std::string signature_name: signature_names)
    {
      output << ',' << signatures.get_signature(filename.c_str(), signature_name.c_str());
    }

    output << std::endl;
  }

  output.flush();
  output.close();
}



void CUDASignatures::compute(char **directories, int count)
{
  reset_directory_tracker(directories, count);
  while(read_next_batch())
    compute_signatures_on_cuda();
}



bool CUDASignatures::supported_format(char *filename)
{
  if (strstr(filename, ".tif") || strstr(filename, ".TIF"))
    return true;
  return false;
}



void CUDASignatures::reset_directory_tracker(char **directories, int count)
{
  directory_tracker.directories = directories;
  directory_tracker.current_dir = 0;
  directory_tracker.count       = count;
  directory_tracker.opened_dir  = 0;
}



bool CUDASignatures::read_next_batch()
{
  dirent *entry;
  char filename_buffer[FILENAME_MAX];
  while((entry = read_next_entry()) && !batch_capacity_reached())
  {
    if (entry->d_name[0] == '.')
      continue;

    if (!supported_format(entry->d_name))
      continue;

    join_paths(filename_buffer, directory_tracker.directory, entry->d_name);

    printf("Loading image \"%s\"\n", filename_buffer);
    load_image_matrix(filename_buffer);
  }

  return (image_matrix_count > 0);
}


#define MAX_MATRIX_SIZE 1073741824

bool CUDASignatures::batch_capacity_reached()
{
  long bytes_taken = 0;
  for (int i = 0; i < image_matrix_count; i++)
  {
    bytes_taken += image_matrices[i]->width * image_matrices[i]->height * sizeof(pix_data);
  }
  return (bytes_taken < MAX_MATRIX_SIZE);
}



dirent * CUDASignatures::read_next_entry()
{
  dirent *entry = 0;

  bool done = false;
  while(!done && directory_tracker.current_dir < directory_tracker.count)
  { 
    if(!directory_tracker.opened_dir)
    {
      directory_tracker.opened_dir = opendir(
        directory_tracker.directories[directory_tracker.current_dir]);
    }

    entry = readdir(directory_tracker.opened_dir);
    if (!entry)
    {
      done = true;
    } 
    else
    {
      closedir(directory_tracker.opened_dir);
      directory_tracker.current_dir++;
    }
  }

  return entry;
}



void CUDASignatures::load_image_matrix(char *filename)
{
  if(image_matrix_count >= matrix_container_size)
    double_matrix_container();

  ImageMatrix *matrix = new ImageMatrix();
  matrix->OpenImage(filename);
  image_matrices[image_matrix_count++] = matrix;
}



void CUDASignatures::double_matrix_container()
{
  ImageMatrix **new_container = new ImageMatrix*[matrix_container_size * 2];
  memcpy(new_container, image_matrices, matrix_container_size * sizeof(ImageMatrix*));
  delete [] image_matrices;
  image_matrices = new_container;
  matrix_container_size *= 2;
}



void CUDASignatures::empty_matrix_container()
{
  for(int i = 0; i < image_matrix_count; i++) 
    delete image_matrices[i];
  image_matrix_count = 0;
}



void CUDASignatures::compute_signatures_on_cuda()
{
  printf("Computing signatures for %i images on CUDA:\n", image_matrix_count);

  // Arrange data in RAM
  pix_data **pixels  = new pix_data*[image_matrix_count];
  int *widths  = new int[image_matrix_count];
  int *heights = new int[image_matrix_count];
  int *depths  = new int[image_matrix_count];

  for(int i = 0; i < image_matrix_count; i++)
  {
    widths[i]  = image_matrices[i]->width;
    heights[i] = image_matrices[i]->height;
    depths[i]  = image_matrices[i]->depth;

    int size = widths[i] * heights[i] * depths[i];
    pix_data *pixel_array;
    hipMalloc(&pixel_array, size * sizeof(pix_data));
    hipMemcpy(pixel_array, image_matrices[i]->data, size * sizeof(pix_data), hipMemcpyHostToDevice);
    pixels[i] = pixel_array;
  }

  // Move data from RAM to VRAM
  pix_data **cPixels = 0; 
  int *cWidths = 0, *cHeights = 0, *cDepths = 0;

  hipMalloc(&cPixels,  image_matrix_count * sizeof(pix_data*));
  hipMalloc(&cWidths,  image_matrix_count * sizeof(int));
  hipMalloc(&cHeights, image_matrix_count * sizeof(int));
  hipMalloc(&cDepths,  image_matrix_count * sizeof(int));

  hipMemcpy(cWidths,  widths,  image_matrix_count * sizeof(int),       hipMemcpyHostToDevice);
  hipMemcpy(cHeights, heights, image_matrix_count * sizeof(int),       hipMemcpyHostToDevice);
  hipMemcpy(cDepths,  depths,  image_matrix_count * sizeof(int),       hipMemcpyHostToDevice);
  hipMemcpy(cPixels,  pixels,  image_matrix_count * sizeof(pix_data*), hipMemcpyHostToDevice);

  signatures.clear();

  double *cOutputs = 0;
  hipMalloc(&cOutputs, MAX_OUTPUT_SIZE * image_matrix_count * sizeof(double));

  long *cSizes = 0;
  hipMalloc(&cSizes, image_matrix_count * sizeof(long));

  // Execute the features.
  printf("Performing Zernike texture analysis\n");
  compute_zernike_on_cuda(cPixels, cWidths, cHeights, cDepths, cOutputs, cSizes);

  hipFree(cSizes);
  hipFree(cOutputs);

  for(int i = 0; i < image_matrix_count; i++)
  {
    hipFree(pixels[i]);
  }
  hipFree(cPixels);
  hipFree(cDepths);
  hipFree(cHeights);
  hipFree(cWidths);

  delete [] pixels;
  delete [] depths;
  delete [] heights;
  delete [] widths;

  printf("Signatures computed\n");
  printf("============================================================\n");

  empty_matrix_container();
}



void CUDASignatures::compute_zernike_on_cuda(pix_data **images, int *widths, int *heights, int *depths, double *outputs, long *sizes)
{
  double *d;
  double *r;

  hipMalloc(&d, image_matrix_count * sizeof(double));
  hipMalloc(&r, image_matrix_count * sizeof(double));

  hipMemset(d, 0, image_matrix_count * sizeof(double));
  hipMemset(r, 0, image_matrix_count * sizeof(double));

  zernike<<< 1, image_matrix_count >>>(images, widths, heights, depths, 
                                       d, r, outputs, sizes);

  char buffer[64];
  for(int i = 0; i < image_matrix_count; i++)
  {
    for(int j = 0; j < sizes[i]; j++)
    {
      sprintf(buffer, "Zernike bin %i", j);
      signatures.add_signature(image_matrices[i]->source_file, buffer, outputs[MAX_OUTPUT_SIZE * i + j]);
    }
  }

  hipFree(r);
  hipFree(d);
}
