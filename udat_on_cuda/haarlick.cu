#include "hip/hip_runtime.h"
//---------------------------------------------------------------------------

#pragma hdrstop

#ifndef BORLAND_C
#include <stdlib.h>
#include <stdio.h>
#endif
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "haarlick.h"
#include "CVIPtexture.h"
#include "image_matrix.h"
#include ""
//---------------------------------------------------------------------------
/* haarlick
output -array of double- a pre-allocated array of 28 doubles
*/

__global__ void CUDA_haarlick2d(ImageMatrix *Im, double distance, double *out) {
	const int i = threadIdx.x * blockDim.x + threadIdx.x;
	int a, x, y;
	unsigned char **p_gray;
	TEXTURE *features;
	long angle;
	double min[14], max[14], sum[14];
	double min_value = INF, max_value = -INF;//max_value=pow(2,Im->bits)-1;

	if (distance <= 0) distance = 1;

	p_gray = new unsigned char *[Im->height];
	for (y = 0; y<Im[i].height; y++)
		p_gray[y] = new unsigned char[Im[i].width];
	/* for more than 8 bits - normalize the image to (0,255) range */

	Im[i].BasicStatistics(NULL, NULL, NULL, &min_value, &max_value, NULL, 0);
	for (y = 0; y<Im[i].height; y++)
		for (x = 0; x<Im[i].width; x++)
			if (Im[i].bits>8) 
				p_gray[y][x] = (unsigned char)((Im[i].pixel(x, y, 0).intensity - min_value)*(255.0 / (max_value - min_value)));
			else 
				p_gray[y][x] = (unsigned char)(Im[i].pixel(x, y, 0).intensity);

	for (a = 0; a<14; a++)
	{
		min[a] = INF;
		max[a] = -INF;
		sum[a] = 0;
	}

	for (angle = 0; angle <= 135; angle = angle + 45)
	{
		features = Extract_Texture_Features((int)distance, angle, p_gray, Im[i].height, Im[i].width, (int)max_value);
		/*  (1) Angular Second Moment */
		sum[0] += features->ASM;
		if (features->ASM<min[0]) min[0] = features->ASM;
		if (features->ASM>max[0]) max[0] = features->ASM;
		/*  (2) Contrast */
		sum[1] += features->contrast;
		if (features->contrast<min[1]) min[1] = features->contrast;
		if (features->contrast>max[1]) max[1] = features->contrast;
		/*  (3) Correlation */
		sum[2] += features->correlation;
		if (features->correlation<min[2]) min[2] = features->correlation;
		if (features->correlation>max[2]) max[2] = features->correlation;
		/*  (4) Variance */
		sum[3] += features->variance;
		if (features->variance<min[3]) min[3] = features->variance;
		if (features->variance>max[3]) max[3] = features->variance;
		/*  (5) Inverse Diffenence Moment */
		sum[4] += features->IDM;
		if (features->IDM<min[4]) min[4] = features->IDM;
		if (features->IDM>max[4]) max[4] = features->IDM;
		/*  (6) Sum Average */
		sum[5] += features->sum_avg;
		if (features->sum_avg<min[5]) min[5] = features->sum_avg;
		if (features->sum_avg>max[5]) max[5] = features->sum_avg;
		/*  (7) Sum Variance */
		sum[6] += features->sum_var;
		if (features->sum_var<min[6]) min[6] = features->sum_var;
		if (features->sum_var>max[6]) max[6] = features->sum_var;
		/*  (8) Sum Entropy */
		sum[7] += features->sum_entropy;
		if (features->sum_entropy<min[7]) min[7] = features->sum_entropy;
		if (features->sum_entropy>max[7]) max[7] = features->sum_entropy;
		/*  (9) Entropy */
		sum[8] += features->entropy;
		if (features->entropy<min[8]) min[8] = features->entropy;
		if (features->entropy>max[8]) max[8] = features->entropy;
		/* (10) Difference Variance */
		sum[9] += features->diff_var;
		if (features->diff_var<min[9]) min[9] = features->diff_var;
		if (features->diff_var>max[9]) max[9] = features->diff_var;
		/* (11) Diffenence Entropy */
		sum[10] += features->diff_entropy;
		if (features->diff_entropy<min[10]) min[10] = features->diff_entropy;
		if (features->diff_entropy>max[10]) max[10] = features->diff_entropy;
		/* (12) Measure of Correlation 1 */
		sum[11] += features->meas_corr1;
		if (features->meas_corr1<min[11]) min[11] = features->meas_corr1;
		if (features->meas_corr1>max[11]) max[11] = features->meas_corr1;
		/* (13) Measure of Correlation 2 */
		sum[12] += features->meas_corr2;
		if (features->meas_corr2<min[12]) min[12] = features->meas_corr2;
		if (features->meas_corr2>max[12]) max[12] = features->meas_corr2;
		/* (14) Maximal Correlation Coefficient */
		sum[13] += features->max_corr_coef;
		if (features->max_corr_coef<min[13]) min[13] = features->max_corr_coef;
		if (features->max_corr_coef>max[13]) max[13] = features->max_corr_coef;
		free(features);
	}

	for (y = 0; y<Im[i].height; y++)
		delete p_gray[y];
	delete p_gray;

	/* copy the values to the output array in the right output order */
	double temp[28];
	for (a = 0; a<14; a++)
	{
		temp[a] = sum[a] / 4;
		temp[a + 14] = max[a] - min[a];
	}

	out[0] = temp[0];
	out[1] = temp[14];
	out[2] = temp[1];
	out[3] = temp[15];
	out[4] = temp[2];
	out[5] = temp[16];
	out[6] = temp[9];
	out[7] = temp[23];
	out[8] = temp[10];
	out[9] = temp[24];
	out[10] = temp[8];
	out[11] = temp[22];
	out[12] = temp[11];
	out[13] = temp[25];
	out[14] = temp[4];
	out[15] = temp[18];
	out[16] = temp[13];
	out[17] = temp[27];
	out[18] = temp[12];
	out[19] = temp[26];
	out[20] = temp[5];
	out[21] = temp[19];
	out[22] = temp[7];
	out[23] = temp[21];
	out[24] = temp[6];
	out[25] = temp[20];
	out[26] = temp[3];
	out[27] = temp[17];
}

void allocate_haarlick_memory(ImageMatrix *matrix, double distance, double *out) {
	// haarlick computation
	double *d_distance, *d_out;
	ImageMatrix *d_matrix;
	/* removed currently because I believe the device is able to allocate its own memory for variables declared within a kernel function
	TEXTURE *d_features;
	int d_a, d_x, d_y;
	unsigned char **d_p_gray;
	TEXTURE *d_features;
	long d_angle;
	double d_min[14], d_max[14], d_sum[14];
	double d_min_value = INF, d_max_value = -INF;//max_value=pow(2,Im->bits)-1;
	size_t pitch;

	hipMallocPitch((void**)&d_p_gray, &pitch, matrix->width * sizeof(unsigned char), matrix->height);
	hipMalloc((void**)d_a, sizeof(int));
	hipMalloc((void**)d_x, sizeof(int));
	hipMalloc((void**)d_y, sizeof(int));
	hipMalloc((void**)d_angle, sizeof(long));
	hipMalloc((void**)&d_min, sizeof(double));
	hipMalloc((void**)&d_max, sizeof(double));
	hipMalloc((void**)&d_sum, sizeof(double));
	hipMalloc((void**)&d_min_value, sizeof(double));
	hipMalloc((void**)&d_max_value, sizeof(double));
	hipMalloc((void**)&d_features, sizeof(TEXTURE));
	*/

	// Allocate memory for variables on device
	hipMalloc((void**)&d_matrix, sizeof(ImageMatrix));
	hipMalloc((void**)&d_out, sizeof(double));
	hipMalloc((void**)&d_distance, sizeof(double));


	hipMemcpy(d_matrix, matrix, sizeof(ImageMatrix), hipMemcpyHostToDevice);
	hipMemcpy(d_out, out, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_distance, &distance, sizeof(double), hipMemcpyHostToDevice);
	CUDA_haarlick2d<<<1, 1>>>(matrix, distance, out);
	hipMemcpy(out, d_out, sizeof(double), hipMemcpyDeviceToHost);

	hipFree(d_matrix);
	hipFree(d_out);
	hipFree(d_distance);
}


#pragma package(smart_init)