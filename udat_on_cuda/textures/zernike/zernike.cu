#include "hip/hip_runtime.h"
/****************************************************************************/
/*                                                                          */
/*                                                                          */
/*                              mb_Znl.c                                    */
/*                                                                          */
/*                                                                          */
/*                           Michael Boland                                 */
/*                            09 Dec 1998                                   */
/*                                                                          */     
/*  Revisions:                                                              */
/*  9-1-04 Tom Macura <tmacura@nih.gov> modified to make the code ANSI C    */
/*         and work with included complex arithmetic library from           */
/*         Numerical Recepies in C instead of using the system's C++ STL    */
/*         Libraries.                                                       */
/*                                                                          */
/*  1-29-06 Lior Shamir <shamirl (-at-) mail.nih.gov> modified "factorial"  */
/*  to a loop, replaced input structure with ImageMatrix class.             */
/****************************************************************************/


//---------------------------------------------------------------------------

#pragma hdrstop

#include "../../cuda_signatures.h"
#include "zernike.h"

#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/complex.h>
#include ""
#include "cuda_complex.h"

//---------------------------------------------------------------------------

__device__ double factorial(double n)
{ 
  if (n < 0) 
    return 0;

  double ans = 1;
  for (int a = 1; a <= n; a++)
    ans *= a;
  return ans;
}

/* mb_imgmoments
   calculates the moment MXY for IMAGE

*/
__device__ double mb_imgmoments(pix_data *pixels, int width, int height, int x, int y)
{ 
  double xcoord;
  double sum = 0;
  /* Generate a matrix with the y coordinates of each pixel. */
  for (int col = 0; col < width; col++) 
  {
    for (int row = 0; row < height; row++)
    {
       xcoord = pow((double)(col+1), (double)x);
       if (y != 0)
       {  
         if (x == 0) 
           xcoord = pow(double(row + 1), double(y));
         else
           xcoord = pow(double(col + 1), double(y)) * xcoord;
       }
       sum += xcoord * get_pixel(pixels, width, height, col, row, 0).intensity;
    }
  }

  return sum;
}



__device__ void mb_Znl(long n, long l, double *X, double *Y, double *P, int size, double *out_r, double *out_i)
{
  double x, y, p ;   /* individual values of X, Y, P */
  int i,m;
  fcomplex sum;              /* Accumulator for complex moments */
  fcomplex Vnl;              /* Inner sum in Zernike calculations */
  double* preal;             /* Real part of return value */
  double* pimag;             /* Imag part of return value */

  sum = Complex (0.0, 0.0);

  for(i = 0 ; i < size ; i++) {
    x = X[i] ;
    y = Y[i] ;
    p = P[i] ;

    Vnl = Complex (0.0, 0.0);
    for( m = 0; m <= (n-l)/2; m++) {
      double tmp = (pow((double)-1.0,(double)m)) * ( factorial(n-m) ) /
				( factorial(m) * (factorial((n - 2.0*m + l) / 2.0)) *
	  			(factorial((n - 2.0*m - l) / 2.0)) ) *
				( pow( sqrt(x*x + y*y), (double)(n - 2*m)) );

	  Vnl = Cadd (Vnl, RCmul(tmp, Rpolar(1.0, l*atan2(y,x))) );
      /*
       NOTE: This function did not work with the following:
        ...pow((x*x + y*y), (double)(n/2 -m))...
        perhaps pow does not work properly with a non-integer
        second argument.
       'not work' means that the output did not match the 'old'
        Zernike calculation routines.
      */
    }

    /* sum += p * conj(Vnl) ; */
	sum = Cadd(sum, RCmul(p, Conjg(Vnl)));
  }

  /* sum *= (n+1)/3.14159265 ; */
  sum = RCmul((n+1)/3.14159265, sum);


  /* Assign the returned value */
  *out_r = sum.r ;
  *out_i = sum.i ;

}



__device__ thrust::complex<double> Rpolar2(double rho, double theta)
{
  return thrust::complex<double>(
      rho * cos(theta),
      rho * sin(theta)
    );
}


/* mb_Znl
  Zernike moment generating function.  The moment of degree n and
  angular dependence l for the pixels defined by coordinate vectors
  X and Y and intensity vector P.  X, Y, and P must have the same
  length
*/
//__device__ void mb_Znl(long n, long l, double *X, double *Y, double *P, int size, double *out_r, double *out_i)
//{ 
//  // Accumulator for complex moments
//  thrust::complex<double> sum = thrust::complex<double> (0.0, 0.0);
//  for(int i = 0 ; i < size ; i++) 
//  {
//    // Inner sum in Zernike calculations
//    thrust::complex<double> Vnl = thrust::complex<double> (0.0, 0.0);
//    for(int m = 0; m <= (n-l)/2; m++) 
//    {
//      double tmp = pow(double(-1.0), double(m)) * factorial(n-m) /
//				( factorial(m) * factorial((n - 2.0*m + l) / 2.0) * factorial((n - 2.0*m - l) / 2.0) ) *
//				( pow(sqrt(X[i]*X[i] + Y[i]*Y[i]), double(n - 2*m)) );
//
//      Vnl = Vnl + tmp * Rpolar2(1.0, l * atan2(Y[i], X[i])) ;
//      /*
//       NOTE: This function did not work with the following:
//        ...pow((x*x + y*y), (double)(n/2 -m))...
//        perhaps pow does not work properly with a non-integer
//        second argument.
//       'not work' means that the output did not match the 'old'
//        Zernike calculation routines.
//      */
//    }
//
//    sum = sum + P[i] * thrust::conj<double>(Vnl);
//  }
//
//  /* sum *= (n+1)/3.14159265 ; */
//  sum = ((n+1)/3.14159265) * sum;
//
//
//  /* Assign the returned value */
//  *out_r = sum.real() ;
//  *out_i = sum.imag() ;
//}


__global__ void cuda_zernike(CudaImages images, ZernikeData data)
{  
  int th_idx = blockIdx.x * blockDim.x + threadIdx.x;

  int rows, cols;
  if (data.D[th_idx] <= 0) 
    data.D[th_idx] = 15;

  if (data.R[th_idx] <= 0)
  {  
    rows = images.heights[th_idx];
    cols = images.widths[th_idx];
    data.R[th_idx] = rows / 2;
  }

  // Find all non-zero pixel coordinates and values 
  double psum = 0;

  int size = 0;
  for (int y=0; y < rows; y++) 
  {
    for (int x=0; x < cols; x++) 
    {
      pix_data pixel = get_pixel(images.pixels[th_idx], 
                                 images.widths[th_idx], 
                                 images.heights[th_idx], 
                                 x, y, 0);
      if (pixel.intensity != 0)
      {  
        data.Y[th_idx][size] = double(y+1);
        data.X[th_idx][size] = double(x+1);
        data.P[th_idx][size] = double(pixel.intensity);
        psum += double(pixel.intensity);
        size++;
      }
    }
  }

  // Normalize the coordinates to the center of mass and normalize
  // pixel distances using the maximum radius argument (R) 
  double moment10 = mb_imgmoments(images.pixels[th_idx], images.widths[th_idx], images.heights[th_idx], 1, 0);
  double moment00 = mb_imgmoments(images.pixels[th_idx], images.widths[th_idx], images.heights[th_idx], 0, 0);
  double moment01 = mb_imgmoments(images.pixels[th_idx], images.widths[th_idx], images.heights[th_idx], 0, 1);

  int size2 = 0;
  for (int a = 0; a < size; a++)
  { 
    data.X[th_idx][size2] = (data.X[th_idx][a] - moment10/moment00)/data.R[th_idx];
    data.Y[th_idx][size2] = (data.Y[th_idx][a] - moment01/moment00)/data.R[th_idx];
    data.P[th_idx][size2] = data.P[th_idx][a] / psum;

    double squareX = data.X[th_idx][size2] * data.X[th_idx][size2];
    double squareY = data.Y[th_idx][size2] * data.Y[th_idx][size2];
    double radius = sqrt( squareX + squareY );
    if (radius <= 1.0) 
      size2++;
  }

  int size3 = 0;
  for (int n = 0; n <= data.D[th_idx]; n++) 
  {
    for (int l = 0; l <= n; l++) 
    {
      if (((n - l) % 2) == 0)
      {  
        double preal, pimag;
        mb_Znl(n, l, data.X[th_idx], data.Y[th_idx], data.P[th_idx], size2, &preal, &pimag);
        data.zvalues[th_idx][size3++] = fabs(sqrt(preal*preal + pimag*pimag));
      }
    }
  }
  data.output_size[th_idx] = size3;
}



ZernikeData cuda_allocate_zernike_data(const std::vector<ImageMatrix *> &images)
{
  ZernikeData zdata;

  hipMalloc(&zdata.D,    images.size() * sizeof(double));
  hipMemset(zdata.D,  0, images.size() * sizeof(double));

  hipMalloc(&zdata.R,    images.size() * sizeof(double));
  hipMemset(zdata.R,  0, images.size() * sizeof(double));

  double **Y = new double*[images.size()];
  double **X = new double*[images.size()];
  double **P = new double*[images.size()];
  double **xcoords = new double*[images.size()];
  double **zvalues = new double*[images.size()];
  for(int i = 0; i < images.size(); i++)
  {
    long arr_size = images[i]->width * images[i]->height * sizeof(double);
    hipMalloc(&Y[i], arr_size);
    hipMalloc(&X[i], arr_size);
    hipMalloc(&P[i], arr_size);
    hipMalloc(&xcoords[i], arr_size);
    hipMalloc(&zvalues[i], MAX_OUTPUT_SIZE * sizeof(double));
  }

  hipMalloc(&zdata.X, images.size() * sizeof(double *));
  hipMemcpy(zdata.X, X, images.size() * sizeof(double *), hipMemcpyHostToDevice);
  delete [] X;

  hipMalloc(&zdata.Y, images.size() * sizeof(double *));
  hipMemcpy(zdata.Y,Y, images.size() * sizeof(double* ), hipMemcpyHostToDevice);
  delete [] Y;

  hipMalloc(&zdata.P, images.size() * sizeof(double *));
  hipMemcpy(zdata.P, P, images.size() * sizeof(double* ), hipMemcpyHostToDevice);
  delete [] P;

  hipMalloc(&zdata.xcoords, images.size() * sizeof(double *));
  hipMemcpy(zdata.xcoords, xcoords, images.size() * sizeof(double* ), hipMemcpyHostToDevice);
  delete [] xcoords;

  hipMalloc(&zdata.zvalues, images.size() * sizeof(double *));
  hipMemcpy(zdata.zvalues, zvalues, images.size() * sizeof(double* ), hipMemcpyHostToDevice);
  delete [] zvalues;

  hipMalloc(&zdata.output_size, images.size() * sizeof(long));
  hipMemset(zdata.output_size, 0, images.size() * sizeof(long));

  return zdata;
}



std::vector<FileSignatures> cuda_get_zernike_signatures(const std::vector<ImageMatrix *> &images, 
                                                        const ZernikeData &data, int image_count)
{
  long *output_size = new long[image_count];
  hipMemcpy(output_size, data.output_size, image_count * sizeof(long), hipMemcpyDeviceToHost);
  hipError_t error = hipGetLastError();
  if (error != hipSuccess)
    std::cout << error << std::endl
      << hipGetErrorName(error) << std::endl
      << hipGetErrorString(error) << std::endl;

  double **zvalues = new double*[image_count];
  hipMemcpy(zvalues, data.zvalues, image_count * sizeof( double * ), hipMemcpyDeviceToHost);
  for(int i = 0; i < image_count; i++) {
    double *zvals = new double[output_size[i]];
    hipMemcpy(zvals, zvalues[i], output_size[i] * sizeof(double), hipMemcpyDeviceToHost);
    zvalues[i] = zvals;
  }

  std::vector<FileSignatures> file_signatures;
  for(int i = 0; i < image_count; i++) 
  {
    FileSignatures file_signature;
    file_signature.file_name = images[i]->source_file;
    int x = 0;
    int y = 0;
    for(int j = 0; j < output_size[i]; j++) 
    {
      std::stringstream ss;
      ss << "ZernikeMoments Z_" << y << "_" << x;

      Signature signature;
      signature.signature_name = ss.str();
      signature.value = zvalues[i][j];

      file_signature.signatures.push_back(signature);

      if (x >= y)
        x = 1 - (y++ % 2);
      else
        x += 2;
    }
    file_signatures.push_back(file_signature);
  }

  for(int i = 0; i < image_count; i++) {
    delete [] zvalues[i];
  }
  delete [] zvalues;
  delete [] output_size;

  return file_signatures;
}



void cuda_delete_zernike_data(ZernikeData &data, int image_count)
{
  hipFree(&data.D);
  hipFree(&data.R);

  double **Y       = new double*[image_count];
  double **X       = new double*[image_count];
  double **P       = new double*[image_count];
  double **xcoords = new double*[image_count];
  double **zvalues = new double*[image_count];

  hipMemcpy(Y,       data.Y,       image_count * sizeof(double *), hipMemcpyDeviceToHost);
  hipMemcpy(X,       data.X,       image_count * sizeof(double *), hipMemcpyDeviceToHost);
  hipMemcpy(P,       data.P,       image_count * sizeof(double *), hipMemcpyDeviceToHost);
  hipMemcpy(xcoords, data.xcoords, image_count * sizeof(double *), hipMemcpyDeviceToHost);
  hipMemcpy(zvalues, data.zvalues, image_count * sizeof(double *), hipMemcpyDeviceToHost);

  for(int i = 0; i < image_count; i++)
  {
    hipFree(&Y[i]);
    hipFree(&X[i]);
    hipFree(&P[i]);
    hipFree(&xcoords[i]);
    hipFree(&zvalues[i]);
  }

  delete [] Y;
  delete [] X;
  delete [] P;
  delete [] xcoords;
  delete [] zvalues;

  hipFree(data.Y);
  hipFree(data.X);
  hipFree(data.P);
  hipFree(data.xcoords);
  hipFree(data.zvalues);
  hipFree(data.output_size);

  memset(&data, 0, sizeof(ZernikeData));
}



#pragma package(smart_init)


