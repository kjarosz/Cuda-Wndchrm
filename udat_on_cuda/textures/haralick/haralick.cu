#include "hip/hip_runtime.h"
//---------------------------------------------------------------------------

#pragma hdrstop

#include "haralick.h"
#include "../../CVIPtexture.h"
#include "../../image_matrix.h"
#include "../../utils/cuda_utils.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include ""



__global__ void haralick(CudaImages images, HaralickData data) 
// pix_data *pixels, double *distance, double *out, int *height, int *width, int *depth, unsigned short int *bits) 
{
	const int th_idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (data.distance[th_idx] <= 0) 
    data.distance[th_idx] = 1;

  unsigned int pixel_count = images.heights[th_idx] * images.widths[th_idx] * images.depths[th_idx];

	double min_value = INF;
  double max_value = -INF; 
	get_intensity_range(images.pixels[th_idx], pixel_count, &min_value, &max_value);
  normalize_to_8_bits(images.pixels[th_idx], images.widths[th_idx], images.heights[th_idx],
                      images.bits[th_idx], min_value, max_value, data.gray[th_idx]);

	for (int a = 0; a < HARALICK_FEATURE_SIZE; a++)
	{
		data.min[th_idx][a] = INF;
		data.max[th_idx][a] = -INF;
		data.sum[th_idx][a] = 0;
	}

	for (long angle = 0; angle <= 135; angle = angle + 45)
	{
    TEXTURE features;
		Extract_Texture_Features(&features, (int)data.distance[th_idx], angle, data.gray[th_idx], 
                             images.heights[th_idx], images.widths[th_idx], (int)max_value);

		/*  (1) Angular Second Moment */
    assign_feature(features.ASM,           &data.min[th_idx][0], &data.max[th_idx][0], &data.sum[th_idx][0]);

		/*  (2) Contrast */
    assign_feature(features.contrast,      &data.min[th_idx][1], &data.max[th_idx][1], &data.sum[th_idx][1]);

		/*  (3) Correlation */
    assign_feature(features.correlation,   &data.min[th_idx][2], &data.max[th_idx][2], &data.sum[th_idx][2]);

		/*  (4) Variance */
    assign_feature(features.variance,      &data.min[th_idx][3], &data.max[th_idx][3], &data.sum[th_idx][3]);

		/*  (5) Inverse Diffenence Moment */
    assign_feature(features.IDM,           &data.min[th_idx][4], &data.max[th_idx][4], &data.sum[th_idx][4]);

		/*  (6) Sum Average */
    assign_feature(features.sum_avg,       &data.min[th_idx][5], &data.max[th_idx][5], &data.sum[th_idx][5]);

		/*  (7) Sum Variance */
    assign_feature(features.sum_var,       &data.min[th_idx][6], &data.max[th_idx][6], &data.sum[th_idx][6]);

		/*  (8) Sum Entropy */
    assign_feature(features.sum_entropy,   &data.min[th_idx][7], &data.max[th_idx][7], &data.sum[th_idx][7]);

		/*  (9) Entropy */
    assign_feature(features.entropy,       &data.min[th_idx][8], &data.max[th_idx][8], &data.sum[th_idx][8]);

		/* (10) Difference Variance */
    assign_feature(features.diff_var,      &data.min[th_idx][9], &data.max[th_idx][9], &data.sum[th_idx][9]);

		/* (11) Diffenence Entropy */
    assign_feature(features.diff_entropy,  &data.min[th_idx][10], &data.max[th_idx][10], &data.sum[th_idx][10]);

		/* (12) Measure of Correlation 1 */
    assign_feature(features.meas_corr1,    &data.min[th_idx][11], &data.max[th_idx][11], &data.sum[th_idx][11]);

		/* (13) Measure of Correlation 2 */
    assign_feature(features.meas_corr2,    &data.min[th_idx][12], &data.max[th_idx][12], &data.sum[th_idx][12]);

		/* (14) Maximal Correlation Coefficient */
    assign_feature(features.max_corr_coef, &data.min[th_idx][13], &data.max[th_idx][13], &data.sum[th_idx][13]);
	}

	/* copy the values to the output array in the right output order */
	for (unsigned int a = 0; a < HARALICK_FEATURE_SIZE; a++)
	{
		data.out_buffer[th_idx][a]                         = data.sum[th_idx][a] / 4;
		data.out_buffer[th_idx][a + HARALICK_FEATURE_SIZE] = data.max[th_idx][a] - data.min[th_idx][a];
	}

  for (unsigned int a = 0; a < HARALICK_OUT_SIZE; a++) 
    data.out[th_idx][a] = data.out_buffer[th_idx][HARALICK_OUT_MAP[a]];
}



__device__ void get_intensity_range(pix_data *pixels, int pixel_count, double *min, double *max)
{
	double min1     = INF;
  double max1     = -INF;

	/* compute min and max */
	for (long pixel_index = 0; pixel_index < pixel_count; pixel_index++)
	{
		if (pixels[pixel_index].intensity > max1)
			max1 = pixels[pixel_index].intensity;

		if (pixels[pixel_index].intensity < min1)
			min1 = pixels[pixel_index].intensity;
	}

	if (max) *max = max1;
	if (min) *min = min1;
}



__device__ void normalize_to_8_bits(pix_data *image, int width, int height, int bits, 
                                    double min, double max, unsigned char **gray)
{
	// for more than 8 bits - normalize the image to (0,255) range 
	for (int y = 0; y < height; y++) {
		for (int x = 0; x < width; x++) { 
      pix_data pixel = get_pixel(image, x, y, 0, width, height);

			if (bits > 8) 
				gray[y][x] = unsigned char((pixel.intensity - min)*(255.0 / (max - min)));
			else 
				gray[y][x] = unsigned char(pixel.intensity);
    }
  }
}



__device__ inline void assign_feature(float feature, double *min, double *max, double *sum)
{
  *sum += feature;

  if (feature < (*min)) 
    *min = feature;

  if (feature > (*max)) 
    *max = feature;
}



HaralickData cuda_allocate_haralick_data(const std::vector<ImageMatrix *> &images)
{
  HaralickData data;
  memset(&data, 0, sizeof(data));

  hipMalloc(&data.distance, sizeof(double) * images.size());
  hipMemset(data.distance, 0 , sizeof(double) * images.size());

  unsigned char ***th_gray = new unsigned char**[images.size()];
  for(int i = 0; i < images.size(); i++)
  {
    unsigned char **gray = new unsigned char*[images[i]->height];
    for(int j = 0; j < images[i]->height; j++) 
      hipMalloc(&gray[j], sizeof(unsigned char) * images[i]->width);

    hipMalloc(&th_gray[i], sizeof(unsigned char *) * images[i]->height);
    hipMemcpy(th_gray[i], gray, sizeof(unsigned char*) * images[i]->height, hipMemcpyHostToDevice);

    delete [] gray;
  }

  hipMalloc(&data.gray, sizeof(unsigned char **) * images.size());
  hipMemcpy(data.gray, th_gray, sizeof(unsigned char **) * images.size(), hipMemcpyHostToDevice);
  delete [] th_gray;

  cuda_alloc_cube_array<double>(HARALICK_FEATURE_SIZE, images.size(), data.min);
  cuda_alloc_cube_array<double>(HARALICK_FEATURE_SIZE, images.size(), data.max);
  cuda_alloc_cube_array<double>(HARALICK_FEATURE_SIZE, images.size(), data.sum);

  cuda_alloc_cube_array<double>(HARALICK_OUT_SIZE, images.size(), data.out_buffer);
  cuda_alloc_cube_array<double>(HARALICK_OUT_SIZE, images.size(), data.out);
}



std::vector<FileSignatures> cuda_get_haralick_signatures(const std::vector<ImageMatrix *> &images, HaralickData &data)
{
	int outs_size = MAX_OUTPUT_SIZE * image_matrix_count;
  double *outs = new double[MAX_OUTPUT_SIZE * image_matrix_count];

  int   sizes_size = image_matrix_count;
  long *lSizes     = new long[image_matrix_count];

  hipMemcpy(outs, outputs, outs_size * sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(lSizes, sizes, sizes_size * sizeof(long), hipMemcpyDeviceToHost);

  char buffer[64];
  for(int i = 0; i < image_matrix_count; i++)
  {
    for(int j = 0; j < lSizes[i]; j++)
    {
      sprintf(buffer, "Haarlick bin %i", j);
      double value = outs[i * MAX_OUTPUT_SIZE + j];
      signatures.add_signature(buffer, image_matrices[i]->source_file, value);
    }
  }

  delete [] outs;
  delete [] lSizes;
  std::vector<FileSignatures> signatures;
  return signatures;
}



void cuda_delete_haralick_data(const std::vector<ImageMatrix *> &images, HaralickData &data)
{
  hipFree(data.distance);

  // Gray stuff
  unsigned char ***th_gray = new unsigned char**[images.size()];
  hipMemcpy(th_gray, data.gray, sizeof(unsigned char **) * images.size(), hipMemcpyDeviceToHost);
  for(int i = 0; i < images.size(); i++)
  {
    unsigned char ** gray = new unsigned char*[images[i]->width];
    hipMemcpy(gray, th_gray[i], sizeof(unsigned char *) * images[i]->width, hipMemcpyDeviceToHost);
    for(int j = 0; j < images[i]->width; j++)
      hipFree(gray[j]);
    hipFree(th_gray[i]);
    delete [] gray;
  }
  delete [] th_gray;
  hipFree(data.gray);

  cuda_free_multidim_arr<double>(data.min,        images.size());
  cuda_free_multidim_arr<double>(data.max,        images.size());
  cuda_free_multidim_arr<double>(data.sum,        images.size());

  cuda_free_multidim_arr<double>(data.out_buffer, images.size());
  cuda_free_multidim_arr<double>(data.out,        images.size());
}

#pragma package(smart_init)