#include <fstream>
#include <sstream>
#include <iostream>
#include <cstdio>



#include "textures/zernike/zernike.h"
#include "textures/haralick/haralick.h"
#include "utils/DirectoryListing.h"
#include "histogram.h"
#include "cuda_signatures.h"
#include "utils/Utils.h"
#include "utils/cuda_utils.h"



std::vector<ClassSignatures> compute_signatures(char *root_dir, char **directories, int count)
{
  std::vector<ClassSignatures> class_signatures;
  for(int i = 0; i < count; i++)
  {
    std::stringstream class_dir;
    class_dir << root_dir << "\\" << directories[i];

    ClassSignatures signatures = compute_class_signatures(class_dir.str());
    class_signatures.push_back(signatures);
  }
  return class_signatures;
}



ClassSignatures compute_class_signatures(std::string class_dir)
{
  DirectoryListing *directory_listing = new DirectoryListing(class_dir);

  std::vector<ImageMatrix *> images;
  ClassSignatures class_signatures;
  class_signatures.class_name = class_dir;
  while(get_next_batch(directory_listing, images))
    for(FileSignatures signature: compute_signatures_on_cuda(images))
      class_signatures.signatures.push_back(signature);

  delete directory_listing;

  return class_signatures;
}



std::vector<FileSignatures> compute_signatures_on_cuda(std::vector<ImageMatrix *> &images)
{
  std::cout << "Computing signatures for " << images.size() << " images on CUDA:" << std::endl;

  std::vector<FileSignatures> signatures;

  CudaImages cuda_images;
  move_images_to_cuda(images, cuda_images);

  // Execute the features.
  merge_signatures(signatures, compute_zernike_on_cuda(images, cuda_images));
//  compute_haralick_on_cuda(cPixels, cWidths, cHeights, cDepths, cOutputs, cSizes, cBits);
//  compute_histogram_on_cuda(cPixels, cWidths, cHeights, cDepths, cOutputs, cSizes, cBits);


  std::cout << "Signatures computed" << std::endl;
  std::cout << "============================================================" << std::endl;

  return signatures;
}



bool get_next_batch(DirectoryListing *listing, std::vector<ImageMatrix *> &images)
{
  for (ImageMatrix *image: images)
    delete image;
  images.clear();

  try 
  {
    while(!batch_is_full(images))
    {
      std::string filename = listing->next_file();
      if (supported_format(filename.c_str()))
      {
        ImageMatrix *image = load_image_matrix(filename.c_str());
        if (image)
          images.push_back(image);
      }
    }
  } 
  catch( OutOfFilesException &exc) 
  {
    if (images.size() == 0)
      return false;
  }

  return true;
}



bool batch_is_full(std::vector<ImageMatrix *> &images)
{
  long bytes_taken = 0;
  for(ImageMatrix *image: images)
    bytes_taken += image->width * image->height * sizeof(pix_data);
  return bytes_taken >= BATCH_SIZE;
}



bool supported_format(const char *filename)
{
  int period = -1;
  unsigned int len, i;
  len = strlen(filename);
  for(i = len - 1; i > 0; i--) {
    if (filename[i] == '.') {
      period = i;
      break;
    }
  }

  if (period <= 0) 
    return false;

  // TODO Check if this compares the extension correctly.
  if (strstr(filename + period, ".tif") || strstr(filename + period, ".TIF"))
    return true;

  return false;
}



ImageMatrix *load_image_matrix(const char *filename)
{
  ImageMatrix *matrix = new ImageMatrix();
  if(!matrix->OpenImage(filename)) {
    delete matrix;
    matrix = 0;
  }
  return matrix;
}



void move_images_to_cuda(std::vector<ImageMatrix *> &images, CudaImages &cuda_images)
{
  cuda_images.count = images.size();

  // Arrange data in RAM
  int *widths        = new int[cuda_images.count];
  int *heights       = new int[cuda_images.count];
  int *depths        = new int[cuda_images.count];
  int *bits          = new int[cuda_images.count];
  pix_data **pixels  = new pix_data*[cuda_images.count];

  for(int i = 0; i < cuda_images.count; i++)
  {
    widths[i]  = images[i]->width;
    heights[i] = images[i]->height;
    depths[i]  = images[i]->depth;
    bits[i]    = images[i]->bits;
    move_host_to_cuda<pix_data>(images[i]->data, widths[i] * heights[i] * depths[i], pixels[i]);
  }

  // Move data from RAM to VRAM
  move_host_to_cuda<pix_data*>(pixels,  cuda_images.count, cuda_images.pixels);
  move_host_to_cuda<int>      (widths,  cuda_images.count, cuda_images.widths);
  move_host_to_cuda<int>      (heights, cuda_images.count, cuda_images.heights);
  move_host_to_cuda<int>      (depths,  cuda_images.count, cuda_images.depths);
  move_host_to_cuda<int>      (bits,    cuda_images.count, cuda_images.bits);

  delete [] pixels;
  delete [] depths;
  delete [] heights;
  delete [] widths;
  delete [] bits;
}



void delete_cuda_images(CudaImages &cuda_images)
{
  cuda_free_multidim_arr<pix_data>(cuda_images.pixels, cuda_images.count);
  hipFree(cuda_images.depths);
  hipFree(cuda_images.heights);
  hipFree(cuda_images.widths);
  hipFree(cuda_images.bits);

  memset(&cuda_images, 0, sizeof(CudaImages));
}




std::vector<FileSignatures> &merge_signatures(std::vector<FileSignatures> &dst,
                                              std::vector<FileSignatures> &src)
{
  for(FileSignatures src_signatures: src)
  {
    bool found = false;
    for(int i = 0; i < dst.size() && !found; i++)
    {
      if (src_signatures.file_name == dst[i].file_name) {
        found = true;
        for (Signature sig: src_signatures.signatures)
          dst[i].signatures.push_back(sig);
      }
    }
    if(!found)
      dst.push_back(src_signatures);
  }
  return dst;
}



std::vector<FileSignatures> compute_zernike_on_cuda(const std::vector<ImageMatrix *> &images, CudaImages &cuda_images)
{
  std::cout << "Performing Zernike texture analysis" << std::endl;

  ZernikeData zernike_data = cuda_allocate_zernike_data(images);
  cuda_zernike<<< 1, cuda_images.count >>>(cuda_images, zernike_data);
  hipError_t sync_error = hipGetLastError();
  hipError_t async_error = hipDeviceSynchronize();

  std::vector<FileSignatures> signatures;
  if(sync_error == hipSuccess && async_error == hipSuccess)
  {
    signatures = cuda_get_zernike_signatures(images, zernike_data, cuda_images.count);
  } 
  else 
  {
    if (sync_error != hipSuccess)
      print_cuda_error(sync_error, "Synchronous CUDA error occurred");

    if (async_error != hipSuccess)
      print_cuda_error(async_error, "Asynchronous CUDA error occurred");
  }
  cuda_delete_zernike_data(zernike_data, cuda_images.count);
  return signatures;
}



std::vector<FileSignatures> compute_haralick_on_cuda(const std::vector<ImageMatrix *> &images, CudaImages &cuda_images)
{
  printf("Performing Haralick texture analysis\n");

  HaralickData haralick_data = cuda_allocate_haralick_data(images);
	cuda_haralick<<< 1, cuda_images.count >>>(cuda_images, haralick_data);
  hipError_t sync_status = hipGetLastError();
  hipError_t async_status = hipDeviceSynchronize();

  std::vector<FileSignatures> signatures;
  if(sync_status == hipSuccess && async_status == hipSuccess)
  {
    signatures = cuda_get_haralick_signatures(images, haralick_data);
  }
  else
  {
    if (sync_status != hipSuccess)
      print_cuda_error(sync_status, "Synchronous CUDA error occurred");

    if (async_status != hipSuccess)
      print_cuda_error(async_status, "Asynchronous CUDA error occurred");
  }
  cuda_delete_haralick_data(images, haralick_data);
  return signatures;
//	int outs_size = MAX_OUTPUT_SIZE * image_matrix_count;
//  double *outs = new double[MAX_OUTPUT_SIZE * image_matrix_count];
//
//  int   sizes_size = image_matrix_count;
//  long *lSizes     = new long[image_matrix_count];
//
//  hipMemcpy(outs, outputs, outs_size * sizeof(double), hipMemcpyDeviceToHost);
//  hipMemcpy(lSizes, sizes, sizes_size * sizeof(long), hipMemcpyDeviceToHost);
//
//  char buffer[64];
//  for(int i = 0; i < image_matrix_count; i++)
//  {
//    for(int j = 0; j < lSizes[i]; j++)
//    {
//      sprintf(buffer, "Haarlick bin %i", j);
//      double value = outs[i * MAX_OUTPUT_SIZE + j];
//      signatures.add_signature(buffer, image_matrices[i]->source_file, value);
//    }
//  }
//
//  delete [] outs;
//  delete [] lSizes;
}

//void CUDASignatures::compute_histogram_on_cuda(pix_data **images, int *widths, int *heights, int *depths, double *outputs, long *sizes, int *bits)
//{
//  printf("Performing Multiscale Histogram analysis\n");
//
//	multiscalehistogram<<< 1, image_matrix_count >>>(images, outputs, widths, heights, depths, bits);
//
//  int outs_size = MAX_OUTPUT_SIZE * image_matrix_count;
//  double *outs = new double[MAX_OUTPUT_SIZE * image_matrix_count];
//
//  int   sizes_size = image_matrix_count;
//  long *lSizes     = new long[image_matrix_count];
//
//  hipMemcpy(outs, outputs, outs_size * sizeof(double), hipMemcpyDeviceToHost);
//  hipMemcpy(lSizes, sizes, sizes_size * sizeof(long), hipMemcpyDeviceToHost);
//
//  char buffer[64];
//  for(int i = 0; i < image_matrix_count; i++)
//  {
//    for(int j = 0; j < lSizes[i]; j++)
//    {
//      sprintf(buffer, "Multiscale Histogram bin %i", j);
//      double value = outs[i * MAX_OUTPUT_SIZE + j];
//      signatures.add_signature(buffer, image_matrices[i]->source_file, value);
//    }
//  }
//
//  delete [] outs;
//  delete [] lSizes;
//}



void save_signatures(std::vector<ClassSignatures> &class_signatures, char *directory)
{
  char buffer[FILENAME_MAX];
  strcpy(buffer, directory);
  strcat(buffer, "\\");
  strcat(buffer, "output.csv");

  std::ofstream output;
  try {
    output.open(buffer);
  } catch (std::system_error &e) {
    std::cout << "Failed to open file \"" << buffer << "\"" << std::endl
              << e.what() << std::endl;
    return;
  }

  std::vector<std::string>         signatures;
  std::vector<std::string>         filenames;
  std::vector<std::vector<double>> cube;
  for(ClassSignatures class_signature: class_signatures)
  {
    for(FileSignatures file_signatures: class_signature.signatures)
    {
      filenames.push_back(file_signatures.file_name);

      std::vector<double> row(signatures.size());
      row.resize(signatures.size());
      for(Signature signature: file_signatures.signatures)
      {
        int i = find_in_vector(signatures, signature.signature_name);
        if (i != -1)
          row[i] = signature.value;
        else
        {
          row.push_back(signature.value);
          signatures.push_back(signature.signature_name);
        }
      }
      cube.push_back(row);
    }
  }

  output << "\"Filename\"";
  for(std::string signature: signatures)
    output << ",\"" << signature << "\"";
  output << std::endl;

  for(int i = 0; i < filenames.size(); i++) {
    output << "\"" << filenames[i] << "\"";

    for(int j = 0; j < cube[i].size(); j++)
      output << ",\"" << cube[i][j] << "\"";

    output << std::endl;
  }

  output.flush();
  output.close();
}



int find_in_vector(std::vector<std::string> &vector, std::string value)
{
  for(int i = 0; i < vector.size(); i++)
  {
    if (value.compare(vector[i]) == 0)
      return i;
  }
  return -1;
}